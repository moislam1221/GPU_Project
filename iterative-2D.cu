#include "hip/hip_runtime.h"
#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

enum method_type { JACOBI, GS, SOR };

template <typename method_type>
__host__ __device__
double iterativeOperation(const double leftMatrix, const double centerMatrix, const double rightMatrix, const double topMatrix, const double bottomMatrix, double leftX, double centerX, double rightX, double topX, double bottomX, const double centerRhs, int gridPoint, method_type method)
{
    double gridValue = centerX;
    switch(method)
    {
        case JACOBI:
	    return gridValue = (centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix;
	case GS:
	    if (gridPoint % 2 == 1) {
	        return gridValue = (centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix;
	    }
	case SOR:
	    double relaxation = 1.9939;
	    if (gridPoint % 2 == 1) {
	        return gridValue = relaxation*((centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix) + (1.0-relaxation)*centerX;
	    }
    }
    return gridValue;
}


template <typename method_type>
__host__ __device__
double iterativeOperation2(const double leftMatrix, const double centerMatrix, const double rightMatrix, const double topMatrix, const double bottomMatrix, double leftX, double centerX, double rightX, double topX, double bottomX, const double centerRhs, int gridPoint, method_type method)
{
    double gridValue = centerX;
    switch(method)
    {
	case JACOBI:	
	    return gridValue = (centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix;
	case GS:
	    if (gridPoint % 2 == 0) {
	        return gridValue = (centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix;
	    }
	case SOR:
	    double relaxation = 1.9939;
	    if (gridPoint % 2 == 0) {
	        return gridValue = relaxation*((centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix) + (1.0-relaxation)*centerX;
	    }
    }
    return gridValue;
}

__device__ __host__
double jacobiGrid(const double leftMatrix, const double centerMatrix, const double rightMatrix, 
		  const double topMatrix, const double bottomMatrix, 
		  const double leftX, double centerX, const double rightX, const double topX, const double bottomX,
		  const double centerRhs)
{
    return (centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX))
         / centerMatrix;
}

__device__ __host__
double RBGSGrid(const double leftMatrix, const double centerMatrix, const double rightMatrix,
		const double topMatrix, const double bottomMatrix,
		const double leftX, double centerX, const double rightX, const double topX, const double bottomX,
		const double centerRhs, const int gridPoint)
{  
    
    // Update all points of a certain parity (i.e. update red, keep black the same)
    if (gridPoint % 2 == 1)
    {
    	return (centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX))
	 / centerMatrix;
    }
    else
    {
	return centerX;
    }
}

__device__ __host__
double SORGrid(const double leftMatrix, const double centerMatrix, const double rightMatrix,
	       const double topMatrix, const double bottomMatrix, 
	       const double leftX, double centerX, const double rightX, const double topX, const double bottomX,
	       const double centerRhs, const int gridPoint)
{  
    // Similar to red-black gauss-seidel, but take weighted average of rbgs 
    // value and current centerX value based on relaxation parameter
    // printf("Relaxation is %f\n", relaxation);
    double relaxation = 1.0;
    if (gridPoint % 2 == 1)
    {
    	return relaxation*((centerRhs - (leftMatrix * leftX + rightMatrix * rightX + topMatrix * topX + bottomMatrix * bottomX)) / centerMatrix) + (1.0-relaxation)*centerX;
    }
    else
    {
	return centerX;
    }
}

double normFromRow(double leftMatrix, double centerMatrix, double rightMatrix, double leftX, double centerX, double rightX,  double centerRhs) 
{
    return centerRhs - (leftMatrix*leftX + centerMatrix*centerX + rightMatrix*rightX);
}

double Residual(const double * solution, const double * rhs, const double * leftMatrix, const double * centerMatrix, const double * rightMatrix, int nGrids)
{
    int nDofs = nGrids;
    double residual = 0.0;
    for (int iGrid = 0; iGrid < nDofs; iGrid++) {
        double leftX = (iGrid > 0) ? solution[iGrid - 1] : 0.0f; 
        double centerX = solution[iGrid];
        double rightX = (iGrid < nGrids - 1) ? solution[iGrid + 1] : 0.0f;
        double residualContributionFromRow = normFromRow(leftMatrix[iGrid], centerMatrix[iGrid], rightMatrix[iGrid], leftX, centerX, rightX, rhs[iGrid]);
	residual = residual + residualContributionFromRow * residualContributionFromRow;
	// printf("For gridpoint %d, residual contribution is %f\n", iGrid, residualContributionFromRow);
    }
    residual = sqrt(residual);
    return residual;
}

/*double * readExactSolution(int nGrids)
{
    double exactSolution[nGrids];
    std::ifstream input("exactSolution.txt");
    for (int i = 0; i < nGrids; i++)
    {
        input >> exactSolution[i];
        // printf("Data is %f\n", exactSolution[i]);
    }
    return exactSolution;
}*/

double solutionError(double * solution, double * exactSolution, int nGrids)
{
    double error = 0.0;
    double difference; 
    for (int iGrid = 0; iGrid < nGrids; iGrid++) {
         difference = solution[iGrid] - exactSolution[iGrid];
	 error = error + difference*difference;
    }
    error = sqrt(error);
    return error;
}


double * iterativeCpu(const double * initX, const double * rhs,
                  const double * leftMatrix, const double * centerMatrix,
                  const double * rightMatrix, const double * topMatrix, 
		  const double * bottomMatrix, int nxGrids, int nyGrids,
		  int nIters, int method)
{
    int nDofs = nxGrids * nyGrids;
    double * x0 = new double[nDofs];
    double * x1 = new double[nDofs];
    memcpy(x0, initX, sizeof(double) * nDofs);
    memcpy(x1, initX, sizeof(double)* nDofs);
    for (int iIter = 0; iIter < nIters; ++ iIter) {
        for (int iGrid = 0; iGrid < nDofs; ++iGrid) {
            double leftX = ((iGrid % nxGrids) == 0) ? 0.0f : x0[iGrid - 1];
            double centerX = x0[iGrid];
            double rightX = (((iGrid + 1) % nxGrids) == 0) ? 0.0f : x0[iGrid + 1];
	    double bottomX = (iGrid < nxGrids) ? 0.0f : x0[iGrid - nxGrids];
            double topX = (iGrid < nDofs - nxGrids) ? x0[iGrid + nxGrids] : 0.0f;
	    if (iIter % 2 == 0) {
                x1[iGrid] = iterativeOperation(leftMatrix[iGrid], centerMatrix[iGrid], rightMatrix[iGrid], topMatrix[iGrid], bottomMatrix[iGrid],
				    leftX, centerX, rightX, topX, bottomX, rhs[iGrid], iGrid, method);
	    }
	    else { 
                x1[iGrid] = iterativeOperation2(leftMatrix[iGrid], centerMatrix[iGrid],
                                    rightMatrix[iGrid], topMatrix[iGrid], bottomMatrix[iGrid],
				    leftX, centerX, rightX, topX, bottomX,
                                    rhs[iGrid], iGrid, method);
            }
        }
        double * tmp = x0; x0 = x1; x1 = tmp;
    }
    delete[] x1;
    return x0;
}


__global__
void _iterativeGpuClassicIteration(double * x1, const double * x0, const double * rhs,
                         const double * leftMatrix, const double * centerMatrix,
                         const double * rightMatrix, const double * topMatrix, const double * bottomMatrix,
			 int nxGrids, int nyGrids, int iteration, int method)
{
    int ixGrid = blockIdx.x * blockDim.x + threadIdx.x; // Col
    int iyGrid = blockIdx.y * blockDim.y + threadIdx.y; // Row
    int iGrid = iyGrid * (nxGrids) + ixGrid;
    int nDofs = nxGrids * nyGrids;
    if (iGrid < nDofs) {
        double leftX = (ixGrid == 0) ? 0.0f : x0[iGrid - 1] ;
        double centerX = x0[iGrid];
        double rightX = (ixGrid == nxGrids - 1) ?  0.0f : x0[iGrid + 1];
	double topX = (iyGrid == nyGrids - 1) ? 0.0f : x0[iGrid + nxGrids];
        double bottomX = (iyGrid == 0) ? 0.0f : x0[iGrid - nxGrids];
	if (iteration % 2 == 0) {
            x1[iGrid] = iterativeOperation(leftMatrix[iGrid], centerMatrix[iGrid],
                                    rightMatrix[iGrid], topMatrix[iGrid], bottomMatrix[iGrid],
				    leftX, centerX, rightX, topX, bottomX, rhs[iGrid], iGrid, method);
	}
	else { 
            x1[iGrid] = iterativeOperation2(leftMatrix[iGrid], centerMatrix[iGrid],
                                    rightMatrix[iGrid], topMatrix[iGrid], bottomMatrix[iGrid],
				    leftX, centerX, rightX, topX, bottomX, rhs[iGrid], iGrid, method);
	}
    }
    __syncthreads();
}

double * iterativeGpuClassic(const double * initX, const double * rhs,
                         const double * leftMatrix, const double * centerMatrix,
                         const double * rightMatrix, const double * topMatrix, const double * bottomMatrix,
			 int nxGrids, int nyGrids, int nIters, const int threadsPerBlock, int method)
{
  	
    int nDofs = nxGrids * nyGrids;
    
    // Allocate memory in the CPU for the solution
    double * x0Gpu, * x1Gpu;
    hipMalloc(&x0Gpu, sizeof(double) * nDofs);
    hipMalloc(&x1Gpu, sizeof(double) * nDofs);
   
    // Allocate CPU memory for other variables
    double * rhsGpu, * leftMatrixGpu, * rightMatrixGpu, * centerMatrixGpu, * topMatrixGpu, * bottomMatrixGpu;
    hipMalloc(&rhsGpu, sizeof(double) * nDofs);
    hipMalloc(&leftMatrixGpu, sizeof(double) * nDofs);
    hipMalloc(&centerMatrixGpu, sizeof(double) * nDofs);
    hipMalloc(&rightMatrixGpu, sizeof(double) * nDofs);
    hipMalloc(&topMatrixGpu, sizeof(double) * nDofs);
    hipMalloc(&bottomMatrixGpu, sizeof(double) * nDofs);
    
    // Allocate GPU memory
    hipMemcpy(x0Gpu, initX, sizeof(double) * nDofs, hipMemcpyHostToDevice);
    hipMemcpy(rhsGpu, rhs, sizeof(double) * nDofs, hipMemcpyHostToDevice);
    hipMemcpy(leftMatrixGpu, leftMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(centerMatrixGpu, centerMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(rightMatrixGpu, rightMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(topMatrixGpu, topMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(bottomMatrixGpu, bottomMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);

    // Run the classic iteration for prescribed number of iterations
    // int threadsPerBlock = 16;
    int nxBlocks = (int)ceil(nxGrids / (double)threadsPerBlock);
    int nyBlocks = (int)ceil(nyGrids / (double)threadsPerBlock);

    dim3 grid(nxBlocks, nyBlocks);
    dim3 block(threadsPerBlock, threadsPerBlock);
    for (int iIter = 0; iIter < nIters; ++iIter) {
	// Jacobi iteration on the CPU (used to be <<<nBlocks, threadsPerBlock>>>)
        _iterativeGpuClassicIteration<<<grid, block>>>(
                x1Gpu, x0Gpu, rhsGpu, leftMatrixGpu, centerMatrixGpu,
                rightMatrixGpu, topMatrixGpu, bottomMatrixGpu,  
		nxGrids, nyGrids, iIter, method); 
        double * tmp = x1Gpu; x0Gpu = x1Gpu; x1Gpu = tmp;
    }

    // Write solution from GPU to CPU variable
    double * solution = new double[nDofs];
    hipMemcpy(solution, x0Gpu, sizeof(double) * nDofs,
            hipMemcpyDeviceToHost);

    // Free all memory
    hipFree(x0Gpu);
    hipFree(x1Gpu);
    hipFree(rhsGpu);
    hipFree(leftMatrixGpu);
    hipFree(centerMatrixGpu);
    hipFree(rightMatrixGpu);

    return solution;
}

__device__ 
void __iterativeBlockUpperPyramidalFromShared(
		double * xLeftBlock, double *xRightBlock, double *xTopBlock, double *xBottomBlock, const double *rhsBlock,
		const double * leftMatrixBlock, const double * centerMatrixBlock,
                const double * rightMatrixBlock, const double * topMatrixBlock, const double * bottomMatrixBlock,
	       	int nxGrids, int nyGrids, int iGrid, int method)
{
    extern __shared__ double sharedMemory[];
    double * x0 = sharedMemory, * x1 = sharedMemory + blockDim.x * blockDim.y; 

    int idx = threadIdx.x + blockDim.x * threadIdx.y;
    
    printf("Idx %d, initial solution %f\n", idx, x0[idx]);
    for (int k = 0; k <= blockDim.x/2-1; ++k) {
           printf("Time step %d\n", k); 
        if (threadIdx.x >= k && threadIdx.x <= blockDim.x-k-1 && threadIdx.y >= k && threadIdx.y <= blockDim.y-k-1) {
        
	// Bottom 
        if (threadIdx.y == k)
        {
	    xBottomBlock[threadIdx.x-k+(2*k)*(blockDim.x-(k-1))] = x0[idx];
     	}
	if (threadIdx.y == k + 1)
    	{
            xBottomBlock[threadIdx.x-k+(2*k)*(blockDim.x-k) + blockDim.x] = x0[idx];
    	}

	// Top
        if (threadIdx.y == blockDim.x - 1 - k)
    	{
	    xTopBlock[threadIdx.x-k+(2*k)*(blockDim.x-(k-1))] = x0[idx];
    	}
	if (threadIdx.y == blockDim.x - 2 - k)
    	{
            xTopBlock[threadIdx.x-k+(2*k)*(blockDim.x-k) + blockDim.x] = x0[idx];
      	}
	
        // Left
        if (threadIdx.x == k)
        {
            xLeftBlock[threadIdx.y-k + (2*k)*(blockDim.x-(k-1))] = x0[idx];
        }
        if (threadIdx.x == k + 1)
        {
            xLeftBlock[threadIdx.y-k + (2*k)*(blockDim.x-(k)) + blockDim.x] = x0[idx];
        }

        // Right
        if (threadIdx.x == blockDim.x - 1 - k)
        {
            xRightBlock[threadIdx.y-k + (2*k)*(blockDim.x-(k-1))] = x0[idx];
        }
        if (threadIdx.x == blockDim.x - 2 - k)
        {
            xRightBlock[threadIdx.y-k + (2*k)*(blockDim.x-(k)) + blockDim.x] = x0[idx];
        }    

	}

        if (threadIdx.x > k && threadIdx.x < blockDim.x-k-1 && threadIdx.y > k && threadIdx.y < blockDim.y-k-1) {
	    
	    double leftX = ((iGrid % nxGrids) == 0) ? 0.0f : x0[idx - 1];
            double centerX = x0[idx];
            double rightX = (((iGrid + 1) % nxGrids) == 0) ? 0.0f : x0[idx + 1];
	    double bottomX = (iGrid < nxGrids) ? 0.0f : x0[idx - blockDim.x];
            double topX = (iGrid < nxGrids*(nyGrids-1)) ? x0[idx + blockDim.x] : 0.0f;
            
	    double leftMat = leftMatrixBlock[idx];
            double centerMat = centerMatrixBlock[idx];
            double rightMat = rightMatrixBlock[idx];
      	    double topMat = topMatrixBlock[idx];
            double bottomMat = bottomMatrixBlock[idx];
            double rhs = rhsBlock[idx];
	    
            if (k % 2 == 0) {
                x1[idx] = centerX; /*iterativeOperation(leftMat, centerMat, rightMat, topMat, bottomMat, 
				                     leftX, centerX, rightX, topX, bottomX, rhs, iGrid, method); */
            }
	    else {
	        x1[idx] = centerX;/* iterativeOperation2(leftMat, centerMat, rightMat, topMat, bottomMat,
				                      leftX, centerX, rightX, topX, bottomX, rhs, iGrid, method); */
	    }
        }
        
	__syncthreads();	
    	double * tmp = x1; x1 = x0; x0 = tmp;
    
    } 

    printf("Idx %d, Top %f, Bottom %f, Left %f, Right %f\n", idx, xTopBlock[idx], xBottomBlock[idx], xLeftBlock[idx], xRightBlock[idx]);    
    printf("Idx %d, SharedMemoryValue %f\n", idx, x0[idx]);
    double * tmp = x1; x1 = x0; x0 = tmp;

}

__global__
void _iterativeGpuUpperPyramidal(double * xLeftGpu, double *xRightGpu, double * xTopGpu, double * xBottomGpu,
                             const double * x0Gpu, const double *rhsGpu, 
                             const double * leftMatrixGpu, const double *centerMatrixGpu, const double * rightMatrixGpu, 
			     const double * topMatrixGpu, const double * bottomMatrixGpu, int nxGrids, int nyGrids, int method)
{
    int xShift = blockDim.x * blockIdx.x;
    int yShift = blockDim.y * blockIdx.y;

    int blockShift = xShift + yShift * nxGrids;

    double * xLeftBlock = xLeftGpu + blockShift;
    double * xRightBlock = xRightGpu + blockShift;
    double * xTopBlock = xTopGpu + blockShift;
    double * xBottomBlock = xBottomGpu + blockShift;
    const double * x0Block = x0Gpu + blockShift;
    const double * rhsBlock = rhsGpu + blockShift;
    const double * leftMatrixBlock = leftMatrixGpu + blockShift;
    const double * centerMatrixBlock = centerMatrixGpu + blockShift;
    const double * rightMatrixBlock = rightMatrixGpu + blockShift;
    const double * topMatrixBlock = topMatrixGpu + blockShift;
    const double * bottomMatrixBlock = bottomMatrixGpu + blockShift;

    int idx = threadIdx.x + threadIdx.y * nxGrids;
    int iGrid = blockShift + idx;
    
    extern __shared__ double sharedMemory[];
    sharedMemory[threadIdx.x + threadIdx.y * blockDim.x] = x0Block[threadIdx.x + threadIdx.y * nxGrids];

    __iterativeBlockUpperPyramidalFromShared(xLeftBlock, xRightBlock, xTopBlock, xBottomBlock, rhsBlock,
    		                             leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, topMatrixBlock, bottomMatrixBlock,
					     nxGrids, nyGrids, iGrid, method);
}

__global__       
void _iterativeGpuLongitudinalBridge(double * xLeftGpu, double * xRightGpu, double * xTopGpu, double * xBottomGpu,
                                  double * rhsGpu, double * leftMatrixGpu, double * centerMatrixGpu, double * rightMatrixGpu, 
				  double * topMatrixGpu, double * bottomMatrixGpu
				  int nxGrids, int nyGrids, int method)
{
    Check all of the shifts in the pointers (almost good - need to double check)
    int numSharedElemPerBlock = blockDim.x * (blockDim.x / 2 + 1);
    int blockID =  blockIdx.y * gridDim.x + blockIdx.x;

    int sharedShift = numSharedElemPerBlock * blockID;
    double * xLowerBlock = xBottomGpu + sharedShift;
    double * xUpperBlock = (blockIdx.y == (gridDim.y-1)) ?
                           xBottomGpu + numSharedElemPerBlock * blockIdx.x : 
                           xBottomGpu + sharedShift + gridDim.x * numSharedElemPerBlock;

    int blockShift = (blockDim.x * blockDim.y) * blockID;
    int verticalShift = blockDim.y/2 * nxGrids;
    
    int idx = threadIdx.x + threadIdx.y * nxGrids;
    int iGrid = blockShift + idx + verticalShift;
    iGrid = (iGrid < nDofs) ? iGrid : iGrid - nDofs; ?? - almost there

    double * rhsBlock = rhsGpu + blockShift + verticalShift;
    double * leftMatrixBlock = leftMatrixGpu + blockShift + verticalShift;
    double * centerMatrixBlock = centerMatrixGpu + blockShift + verticalShift;
    double * rightMatrixBlock = rightMatrixGpu + blockShift + verticalShift;
    double * topMatrixBlock = centerMatrixGpu + blockShift + verticalShift;
    double * bottomMatrixBlock = rightMatrixGpu + blockShift + verticalShift;
    
    extern __shared__ double sharedMemory[];
    
    __iterativeBlockLongitudinalBridgeFromShared(xLowerBlock, xUpperBlock, rhsBlock,
                                       leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, topMatrixBlock, bottomMatrixBlock,
				       nxGrids, nyGrids, iGrid, method);  
}

__global__
void __iterativeBlockLongitudinalBridgeFromShared(double * xLowerBlock, double * xUpperBlock, double * rhsBlock,
		                        double * leftMatrixBlock, double * centerMatrixBlock, double * rightMatrixBlock, double * topMatrixBlock, double * bottomMatrixBlock,
                                        int nxGrids, int ny Grids, int iGrid, int method)
{
    // At every step, load xLower and xUpper and fill in values
    for (int k = 0; k < blockDim.x/2-1; --k) 
    {
	if idx >= 2*k(blockDim.x-(k-1)) && idx <= 2*(k+1)*(blockDim.x-k)
	{
	    x0[??] = xUpper[idx]
	    x0[??] = xLower[idx]
        }

	if (k < blockDim.x/2 - 1) 
	{
	    double leftX = x0[idx];
	    double centerX = x0[idx];
	    double rightX = x0[idx];
	    double topX = x0[idx];
            double bottomX = x0[idx]

            double leftMat = leftMatrixBlock[idx];
	    double centerMat = centerMatrixBlock[idx];
	    double rightMat = rightMatrixBlock[idx];
	    double topMat = topMatrixBlock[idx];
	    double bottomMat = bottomMatrixBlock[idx];

	    if (k % 2 == 0) {
                x1[idx] = iterativeOperation(leftMat, centerMat, rightMat, topMat, bottomMat, leftX, centerX, rightX, topX, bottomX,
					     rhs, iGrid, method);
            }
	    else {
                x1[idx] = iterativeOperation2(leftMat, centerMat, rightMat, topMat, bottomMat, leftX, centerX, rightX, topX, bottomX,
					     rhs, iGrid, method);
            }
	}

    }
 
	    

}



/*
__device__ 
void __iterativeBlockLowerTriangleFromShared(
		const double * xLeftBlock, const double *xRightBlock, const double *rhsBlock,
		const double * leftMatrixBlock, const double * centerMatrixBlock,
                const double * rightMatrixBlock, int nGrids, int iGrid, int method)
{
    extern __shared__ double sharedMemory[];
    double * x0 = sharedMemory, * x1 = sharedMemory + blockDim.x;

    int remainder = threadIdx.x % 4;

    if (threadIdx.x != blockDim.x-1) {
        x0[blockDim.x-1-((blockDim.x+threadIdx.x+1)/2) + blockDim.x*(remainder>1)] = xLeftBlock[threadIdx.x];
	x0[(blockDim.x+threadIdx.x+1)/2 + blockDim.x*(remainder>1)] = xRightBlock[threadIdx.x];
    } 

    # pragma unroll
    for (int k = blockDim.x/2; k > 0; --k) {
	if (k < blockDim.x/2) {
	    if (threadIdx.x >= k && threadIdx.x <= blockDim.x-k-1) {
                double leftX = x0[threadIdx.x - 1];
                double centerX = x0[threadIdx.x];
                double rightX = x0[threadIdx.x + 1];
		if (iGrid == 0) {
		    leftX = 0.0f;
		}
		if (iGrid == nGrids-1) {
		    rightX = 0.0f;
		}
		double leftMat = leftMatrixBlock[threadIdx.x];
		double centerMat = centerMatrixBlock[threadIdx.x];
 		double rightMat = rightMatrixBlock[threadIdx.x];
		double rhs = rhsBlock[threadIdx.x];
	        if (k % 2 == 1) {	
	            x1[threadIdx.x] = iterativeOperation(leftMat, centerMat, rightMat, leftX, centerX, rightX, rhs, iGrid, method);
		}
		else {
		    x1[threadIdx.x] = iterativeOperation2(leftMat, centerMat, rightMat, leftX, centerX, rightX, rhs, iGrid, method);
		}
	    }
 	    double * tmp = x1; x1 = x0; x0 = tmp;
        }
	__syncthreads();
    }

    double leftX = (threadIdx.x == 0) ? xLeftBlock[blockDim.x - 1] : x0[threadIdx.x - 1];
    double centerX = x0[threadIdx.x];
    double rightX = (threadIdx.x == blockDim.x-1) ? xRightBlock[blockDim.x - 1] : x0[threadIdx.x + 1];
    if (iGrid == 0) {
       leftX = 0.0;    
    }
    if (iGrid == nGrids-1) {
        rightX = 0.0;
    }
    // The last step! - Should i just perform one of the grid operations
    // The last step of the for loop above uses k = 1 where gridOperation is used, so I'll use gridOperation2 here
    x1[threadIdx.x] = iterativeOperation2(leftMatrixBlock[threadIdx.x],
                                centerMatrixBlock[threadIdx.x],
                                rightMatrixBlock[threadIdx.x],
                                leftX, centerX, rightX, rhsBlock[threadIdx.x], iGrid, method);
    double * tmp = x1; x1 = x0; x0 = tmp; 

}

__global__
void _iterativeGpuLowerTriangle(double * x0Gpu, double *xLeftGpu,
                             double * xRightGpu, double *rhsGpu, 
                             double * leftMatrixGpu, double *centerMatrixGpu,
                             double * rightMatrixGpu, int nGrids, int method)
{
    int blockShift = blockDim.x * blockIdx.x;
    double * xLeftBlock = xLeftGpu + blockShift;
    double * xRightBlock = xRightGpu + blockShift;
    double * x0Block = x0Gpu + blockShift;
    double * rhsBlock = rhsGpu + blockShift;
    double * leftMatrixBlock = leftMatrixGpu + blockShift;
    double * centerMatrixBlock = centerMatrixGpu + blockShift;
    double * rightMatrixBlock = rightMatrixGpu + blockShift;

    int iGrid = blockIdx.x * blockDim.x + threadIdx.x;
    
    extern __shared__ double sharedMemory[];
    
    __iterativeBlockLowerTriangleFromShared(xLeftBlock, xRightBlock, rhsBlock,
                         leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, nGrids, iGrid, method);

    x0Block[threadIdx.x] = sharedMemory[threadIdx.x];

}

__global__       
void _iterativeGpuShiftedDiamond(double * xLeftGpu, double * xRightGpu,
                              double * rhsGpu, 
			      double * leftMatrixGpu, double * centerMatrixGpu,
                              double * rightMatrixGpu, int nGrids, int method)
{

    int blockShift = blockDim.x * blockIdx.x;
    double * xLeftBlock = xRightGpu + blockShift;
    double * xRightBlock = (blockIdx.x == (gridDim.x-1)) ?
                          xLeftGpu : 
                          xLeftGpu + blockShift + blockDim.x;

    int iGrid = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x/2;
    iGrid = (iGrid < nGrids) ? iGrid : threadIdx.x - blockDim.x/2;

    int indexShift = blockDim.x/2;
    double * rhsBlock = rhsGpu + blockShift + indexShift;
    double * leftMatrixBlock = leftMatrixGpu + blockShift + indexShift;
    double * centerMatrixBlock = centerMatrixGpu + blockShift + indexShift;
    double * rightMatrixBlock = rightMatrixGpu + blockShift + indexShift;
    
    extern __shared__ double sharedMemory[];
    
    __iterativeBlockLowerTriangleFromShared(xLeftBlock, xRightBlock, rhsBlock,
                         leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, nGrids, iGrid, method);  

    __iterativeBlockUpperTriangleFromShared(xLeftBlock, xRightBlock, rhsBlock,
                                       leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, nGrids, iGrid, method);

}

__global__
void _iterativeGpuDiamond(double * xLeftGpu, double * xRightGpu,
                       const double * rhsGpu,
		       const double * leftMatrixGpu, const double * centerMatrixGpu,
                       const double * rightMatrixGpu, int nGrids, int method)
{
    int blockShift = blockDim.x * blockIdx.x;
    double * xLeftBlock = xLeftGpu + blockShift;
    double * xRightBlock = xRightGpu + blockShift;

    const double * rhsBlock = rhsGpu + blockShift;
    const double * leftMatrixBlock = leftMatrixGpu;
    const double * centerMatrixBlock = centerMatrixGpu + blockShift;
    const double * rightMatrixBlock = rightMatrixGpu + blockShift;

    int iGrid = blockDim.x * blockIdx.x + threadIdx.x;
    
    extern __shared__ double sharedMemory[];

    __iterativeBlockLowerTriangleFromShared(xLeftBlock, xRightBlock, rhsBlock,
                        leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, nGrids, iGrid, method);
    
    __iterativeBlockUpperTriangleFromShared(xLeftBlock, xRightBlock, rhsBlock,
                                      leftMatrixBlock, centerMatrixBlock, rightMatrixBlock, nGrids, iGrid, method);
}
*/
double * iterativeGpuSwept(const double * initX, const double * rhs,
        const double * leftMatrix, const double * centerMatrix,
        const double * rightMatrix, const double * topMatrix, const double * bottomMatrix,
	int nxGrids, int nyGrids, int nIters, const int threadsPerBlock, const int method)
{     
    // Determine number of threads and blocks 
    const int nxBlocks = (int)ceil(nxGrids / (double)threadsPerBlock);
    const int nyBlocks = (int)ceil(nyGrids / (double)threadsPerBlock);
    const int nDofs = nxGrids * nyGrids;

    dim3 grid(nxBlocks, nyBlocks);
    dim3 block(threadsPerBlock, threadsPerBlock);
    
    // Allocate memory for solution and inputs
    double *xLeftGpu, *xRightGpu, *xTopGpu, *xBottomGpu;
    hipMalloc(&xLeftGpu, sizeof(double) * threadsPerBlock * nxBlocks);
    hipMalloc(&xRightGpu, sizeof(double) * threadsPerBlock * nxBlocks);
    hipMalloc(&xTopGpu, sizeof(double) * threadsPerBlock * nxBlocks);
    hipMalloc(&xBottomGpu, sizeof(double) * threadsPerBlock * nxBlocks);
    double * x0Gpu, * rhsGpu, * leftMatrixGpu, * rightMatrixGpu, * centerMatrixGpu, * topMatrixGpu, * bottomMatrixGpu;
    hipMalloc(&x0Gpu, sizeof(double) * (nDofs + threadsPerBlock/2));
    hipMalloc(&rhsGpu, sizeof(double) * (nDofs + threadsPerBlock/2));
    hipMalloc(&leftMatrixGpu, sizeof(double) * (nDofs + threadsPerBlock/2));
    hipMalloc(&centerMatrixGpu, sizeof(double) * (nDofs + threadsPerBlock/2));
    hipMalloc(&rightMatrixGpu, sizeof(double) * (nDofs + threadsPerBlock/2));
    hipMalloc(&topMatrixGpu, sizeof(double) * (nDofs + threadsPerBlock/2));
    hipMalloc(&bottomMatrixGpu, sizeof(double) * (nDofs + threadsPerBlock/2));

    // Allocate memory in the GPU
    hipMemcpy(x0Gpu, initX, sizeof(double) * nDofs, hipMemcpyHostToDevice);
    hipMemcpy(rhsGpu, rhs, sizeof(double) * nDofs, hipMemcpyHostToDevice);
    hipMemcpy(leftMatrixGpu, leftMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(centerMatrixGpu, centerMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(rightMatrixGpu, rightMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(topMatrixGpu, topMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);
    hipMemcpy(bottomMatrixGpu, bottomMatrix, sizeof(double) * nDofs,
            hipMemcpyHostToDevice);

    // Allocate a bit more memory to avoid memcpy within shifted kernels
    /*hipMemcpy(x0Gpu + nGrids, initX, sizeof(double) * threadsPerBlock/2, hipMemcpyHostToDevice);
    hipMemcpy(rhsGpu + nGrids, rhs, sizeof(double) * threadsPerBlock/2, hipMemcpyHostToDevice);
    hipMemcpy(leftMatrixGpu + nGrids, leftMatrix, sizeof(double) * threadsPerBlock/2,
            hipMemcpyHostToDevice);
    hipMemcpy(centerMatrixGpu + nGrids, centerMatrix, sizeof(double) * threadsPerBlock/2,
            hipMemcpyHostToDevice);
    hipMemcpy(rightMatrixGpu + nGrids, rightMatrix, sizeof(double) * threadsPerBlock/2,
            hipMemcpyHostToDevice);
    */

    int sharedFloatsPerBlock = threadsPerBlock * threadsPerBlock * 2;

/*    double residualSwept;
    double nCycles = nIters / threadsPerBlock;
    double * currentSolution = new double[nGrids];
    std::ofstream residuals;
    residuals.open("dummy.txt",std::ios_base::app);
    
    for (int i = 0; i < nCycles; i++) {
        _iterativeGpuUpperTriangle <<<nBlocks, threadsPerBlock, sizeof(double) * sharedFloatsPerBlock>>> (xLeftGpu, xRightGpu, x0Gpu, rhsGpu, leftMatrixGpu, centerMatrixGpu, rightMatrixGpu, nGrids, method);
	_iterativeGpuShiftedDiamond <<<nBlocks, threadsPerBlock, sizeof(double) * sharedFloatsPerBlock>>> (xLeftGpu, xRightGpu, rhsGpu, leftMatrixGpu, centerMatrixGpu, rightMatrixGpu, nGrids, method);
	_iterativeGpuLowerTriangle <<<nBlocks, threadsPerBlock, sizeof(double) * sharedFloatsPerBlock>>> (x0Gpu, xLeftGpu, xRightGpu, rhsGpu, leftMatrixGpu, centerMatrixGpu, rightMatrixGpu, nGrids, method);
        hipMemcpy(currentSolution, x0Gpu, sizeof(double) * nGrids,
            hipMemcpyDeviceToHost);
        residualSwept = Residual(currentSolution, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
        residuals << nGrids << "\t" << threadsPerBlock << "\t" << i*threadsPerBlock << "\t" << residualSwept << "\n";
    }
   
    residuals.close();
*/
    _iterativeGpuUpperPyramidal <<<grid, block,
        sizeof(double) * sharedFloatsPerBlock>>>(
                xLeftGpu, xRightGpu, xTopGpu, xBottomGpu,
                x0Gpu, rhsGpu, leftMatrixGpu, centerMatrixGpu,
                rightMatrixGpu, topMatrixGpu, bottomMatrixGpu, 
		nxGrids, nyGrids, method);
    _iterativeGpuLongitudinalBridge <<<grid, block,
            sizeof(double) * sharedFloatsPerBlock>>>(
                    xLeftGpu, xRightGpu, xTopGpu, xBottomGpu,
                    rhsGpu, leftMatrixGpu, centerMatrixGpu,
                    rightMatrixGpu, topMatrixGpu, bottomMatrixGpu,
		    nxGrids, nyGrids, method);
/*
    for (int i = 0; i < nIters/threadsPerBlock-1; i++) {
    _iterativeGpuDiamond <<<grid, block,
                sizeof(double) * sharedFloatsPerBlock>>>(
                        xLeftGpu, xRightGpu,
                        rhsGpu, leftMatrixGpu, centerMatrixGpu,
                        rightMatrixGpu, nGrids, method); 
    _iterativeGpuShiftedDiamond <<<grid, block,
            sizeof(double) * sharedFloatsPerBlock>>>(
                    xLeftGpu, xRightGpu,
                    rhsGpu, leftMatrixGpu, centerMatrixGpu,
                    rightMatrixGpu, nGrids, method); 
    }

    _iterativeGpuLowerTriangle <<<grid, block,
                sizeof(double) * sharedFloatsPerBlock>>>(
                        x0Gpu, xLeftGpu, xRightGpu,
                        rhsGpu, leftMatrixGpu, centerMatrixGpu,
                        rightMatrixGpu, nGrids, method); 
*/
    double * solution = new double[nDofs];
    hipMemcpy(solution, x0Gpu, sizeof(double) * nDofs,
            hipMemcpyDeviceToHost);

    hipFree(x0Gpu);
    hipFree(xLeftGpu);
    hipFree(xRightGpu);
    hipFree(rhsGpu);
    hipFree(leftMatrixGpu);
    hipFree(centerMatrixGpu);
    hipFree(rightMatrixGpu);

    return solution;
}

int main(int argc, char *argv[])
{
    // Ask user for inputs
    const int nxGrids = atoi(argv[1]); 
    const int nyGrids = atoi(argv[2]); 
    const int threadsPerBlock = atoi(argv[3]); 
    const int nIters = atoi(argv[4]);

    method_type method = GS;

    int nDofs = nxGrids * nyGrids;
    
    // Declare arrays and population with values for Poisson equation
    double * initX = new double[nDofs];
    double * rhs = new double[nDofs];
    double * leftMatrix = new double[nDofs];
    double * centerMatrix = new double[nDofs];
    double * rightMatrix = new double[nDofs];
    double * bottomMatrix = new double[nDofs];
    double * topMatrix = new double[nDofs];
    
    double dx = 1.0f / (nxGrids + 1);
    double dy = 1.0f / (nyGrids + 1);

    for (int iGrid = 0; iGrid < nDofs; ++iGrid) {
        initX[iGrid] = (double)iGrid; 
        rhs[iGrid] = 1.0f;
        leftMatrix[iGrid] = -1.0f / (dx * dx);
        centerMatrix[iGrid] = 2.0f / (dx * dx) + 2.0f / (dy * dy);
        rightMatrix[iGrid] = -1.0f / (dx * dx);
	bottomMatrix[iGrid] = -1.0f / (dy * dy);
	topMatrix[iGrid] = -1.0f / (dy * dy);
    }

    // hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    // Run the CPU Implementation and measure the time required
    clock_t cpuStartTime = clock();
    double * solutionCpu = iterativeCpu(initX, rhs, leftMatrix, centerMatrix,
                                    rightMatrix, topMatrix, bottomMatrix, nxGrids, nyGrids, nIters, method);
    clock_t cpuEndTime = clock();
    double cpuTime = (cpuEndTime - cpuStartTime) / (double) CLOCKS_PER_SEC;

    // Run the Classic GPU Implementation and measure the time required
    hipEvent_t startClassic, stopClassic;
    float timeClassic;
    hipEventCreate( &startClassic );
    hipEventCreate( &stopClassic );
    hipEventRecord(startClassic, 0);
    double * solutionGpuClassic = iterativeGpuClassic(initX, rhs, leftMatrix, centerMatrix,
                                                      rightMatrix, topMatrix, bottomMatrix, nxGrids, nyGrids, nIters, threadsPerBlock, method);
    hipEventRecord(stopClassic, 0);
    hipEventSynchronize(stopClassic);
    hipEventElapsedTime(&timeClassic, startClassic, stopClassic);

    // Run the Swept GPU Implementation and measure the time required
    hipEvent_t startSwept, stopSwept;
    float timeSwept;
    hipEventCreate( &startSwept );
    hipEventCreate( &stopSwept );
    hipEventRecord( startSwept, 0);
    double * solutionGpuSwept = iterativeGpuSwept(initX, rhs, leftMatrix, centerMatrix,
                                                  rightMatrix, topMatrix, bottomMatrix, nxGrids, nyGrids, nIters, threadsPerBlock, method);
    hipEventRecord(stopSwept, 0);
    hipEventSynchronize(stopSwept);
    hipEventElapsedTime(&timeSwept, startSwept, stopSwept);
    
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of total grid points: %d\n", nDofs);
    printf("Number of grid points in x-direction: %d\n", nxGrids);
    printf("Number of grid points in y-direction: %d\n", nyGrids);
    printf("Threads Per Block in each direction: %d\n", threadsPerBlock);
    printf("Method used: %d\n", method);
    printf("Number of Iterations performed: %d\n", nIters);
    printf("\n");

    // Print out results to the screen, notify if any GPU Classic or Swept values differ significantly
    for (int iGrid = 0; iGrid < nDofs; ++iGrid) {
        printf("%d %f %f %f \n",iGrid, solutionCpu[iGrid],
                             solutionGpuClassic[iGrid],
                             solutionGpuSwept[iGrid]); 
	//assert(solutionGpuClassic[iGrid] == solutionGpuSwept[iGrid]);
	// if (abs(solutionGpuClassic[iGrid] - solutionGpuSwept[iGrid]) > 1e-2) {
	//    printf("For grid point %d, Classic and Swept give %f and %f respectively\n", iGrid, solutionGpuClassic[iGrid], solutionGpuSwept[iGrid]);
	// }
    }

    // Print out time for cpu, classic gpu, and swept gpu approaches
//    double cpuTimePerIteration = (cpuTime / nIters) * 1e3;
//    double classicTimePerIteration = timeClassic / nIters;
//    double sweptTimePerIteration = timeSwept / nIters;
//    double timeMultiplier = classicTimePerIteration / sweptTimePerIteration;
//    printf("Time needed for the CPU (per iteration): %f ms\n", cpuTimePerIteration);
//    printf("Time needed for the Classic GPU (per iteration) is %f ms\n", classicTimePerIteration);
//    printf("Time needed for the Swept GPU (per iteration): %f ms\n", sweptTimePerIteration); 

    // Compute the residual of the resulting solution (|b-Ax|)
    //double residualClassic = Residual(solutionGpuClassic, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    //double residualSwept = Residual(solutionGpuSwept, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
//    printf("Residual of the converged solution is %f\n", residualSwept);
//    printf("Residual of Classic result is %f\n", residualClassic); 
  
    // Save residual to a file
    /* std::ofstream residuals;
    residuals.open("residual-gs.txt",std::ios_base::app);
    residuals << nGrids << "\t" << threadsPerBlock << "\t" << nIters << "\t" << residualSwept << "\n";
    residuals.close(); */

    // Save Results to a file "N tpb Iterations CPUTime/perstep ClassicTime/perstep SweptTime/perStep ClassicTime/SweptTime"
    std::ofstream timings;
    timings.open("time.txt",std::ios_base::app);
//    timings << nxGrids << "\t" << nyGrids << "\t" << threadsPerBlock << "\t" << nIters << "\t" << cpuTimePerIteration << "\t" << classicTimePerIteration << "\t" << sweptTimePerIteration << "\t" << timeMultiplier << "\n";
    timings.close();

    // Free memory
    hipEventDestroy(startClassic);
    hipEventDestroy(startSwept);
    delete[] initX;
    delete[] rhs;
    delete[] leftMatrix;
    delete[] centerMatrix;
    delete[] rightMatrix;
    delete[] solutionCpu;
    delete[] solutionGpuClassic;
}
