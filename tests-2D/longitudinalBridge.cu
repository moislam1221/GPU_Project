#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

#include "upperPyramidal.h"
#include "longitudinalBridge.h"

int main()
{
    // INPUTS
    int nxGrids = 4;
    int nyGrids = 4;
    int threadsPerBlock = 4;
   
    // SETTING GRID, BLOCK, THREAD INFORMATION 
    int nxBlocks = nxGrids / threadsPerBlock;
    int nyBlocks = nyGrids / threadsPerBlock;
    dim3 grid(nxBlocks, nyBlocks);
    dim3 block(threadsPerBlock, threadsPerBlock);
    int sharedMemory = threadsPerBlock * threadsPerBlock * 2 * sizeof(double);
    
    // PANEL ARRAY SIZES
    int numBridgeElemPerBlock = 2 * threadsPerBlock/2 * (threadsPerBlock/2 + 1);
    int numBridgeElemTotal = nxBlocks * nyBlocks * numBridgeElemPerBlock;

    // OTHER PARAMETERS
    int dx = 1.0/ (nxGrids + 1);
    int dy = 1.0/ (nyGrids + 1);
    int nDofs = nxGrids * nyGrids;
    
    // INT AND POINTER FOR UNUSED PARAMETERS    
    int notUsedInt;
    double * notUsed;

    // INITIALIZATION
    double * initX = new double[nDofs];
    double * xLeftCpu = new double[numBridgeElemTotal];
    double * xRightCpu = new double[numBridgeElemTotal];
    double * xTopCpu = new double[numBridgeElemTotal];
    double * xBottomCpu = new double[numBridgeElemTotal];

    for (int iGrid = 0; iGrid < nDofs; iGrid++) 
    {
        initX[iGrid] = iGrid;
    }

    // ALLOCATE SOLUTION MEMORY - CPU AND GPU
    double * x0Gpu;
    hipMalloc(&x0Gpu, sizeof(double) * (nDofs));
    hipMemcpy(x0Gpu, initX, sizeof(double) * nDofs, hipMemcpyHostToDevice);
    
    // ALLOCATE PANEL ARRAY MEMORY
    double *xLeftGpu, *xRightGpu, *xTopGpu, *xBottomGpu;
    hipMalloc(&xLeftGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xRightGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xTopGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xBottomGpu, sizeof(double) * numBridgeElemTotal);
    
    // APPLY UPPER TRIANGULAR METHOD
    _iterativeGpuUpperPyramidal <<<grid, block, sharedMemory>>> (xLeftGpu, xRightGpu, xTopGpu, xBottomGpu, x0Gpu, notUsed,
		   						 notUsed, notUsed, notUsed, notUsed, notUsed, nxGrids, nyGrids, notUsedInt);
   
    // INITIALIZATION OF NEW BRIDGE ARRAYS
    double * xNorthCpu = new double[numBridgeElemTotal];
    double * xSouthCpu = new double[numBridgeElemTotal];
    double * xEastCpu = new double[numBridgeElemTotal];
    double * xWestCpu = new double[numBridgeElemTotal];

    // ALLOCATE PANEL ARRAY MEMORY
    double *xNorthGpu, *xSouthGpu, *xEastGpu, *xWestGpu;
    hipMalloc(&xNorthGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xSouthGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xEastGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xWestGpu, sizeof(double) * numBridgeElemTotal);
    
    // APPLY LONGITUDINAL BRIDGE METHOD
    _iterativeGpuLongitudinalBridge <<<grid, block, sharedMemory>>> (xTopGpu, xBottomGpu, xEastGpu, xWestGpu, x0Gpu, 
		                                                     notUsed, notUsed, notUsed, notUsed, notUsed, notUsed,
				                                     nxGrids, nyGrids, notUsedInt);

    // COPY RESULTS TO CPU
    //hipMemcpy(xNorthCpu, xNorthGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);    
    //hipMemcpy(xSouthCpu, xSouthGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);    
    hipMemcpy(xEastCpu, xEastGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);    
    hipMemcpy(xWestCpu, xWestGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);    

    // PRINT RESULTS
    for (int iGrid = 0; iGrid < numBridgeElemTotal; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid << " xEast " << xEastCpu[iGrid] << " xWest " << xWestCpu[iGrid] << std::endl;
    }

    // CLEAN UP
    hipFree(xLeftGpu);
    hipFree(xRightGpu);
    hipFree(xTopGpu);
    hipFree(xBottomGpu);
    delete[] initX;
    delete[] xLeftCpu;
    delete[] xRightCpu;
    delete[] xTopCpu;
    delete[] xBottomCpu;    
}
