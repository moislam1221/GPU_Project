#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

#include "iterative-2D-domain-decomposition-gpu.h"
#include "helper.h"

int main(int argc, char *argv[])
{
    // INPUTS
    int nxGrids = 64;
    int nyGrids = 64;
    int subdomainLength = 32; 
    int threadsPerBlock = 32;

    int cycles = 100;
    int num_JacobiIters = 100; 

    printf("Cycles: %d, Jacobi Iterations: %d\n", cycles, num_JacobiIters);

    int method = 0;
   
    float dx = 1.0/ (nxGrids - 1);
    float dy = 1.0/ (nyGrids - 1);
    int nDofs = nxGrids * nyGrids;
    
    // INITIALIZATION
    float * initX = new float[nDofs];
    float * rhs = new float[nDofs];
    float * solutionDDGPU = new float[nDofs];

    // INITIAL SOLUTION
    int dof;
    for (int iGrid = 0; iGrid < nyGrids; iGrid++) { 
        for (int jGrid = 0; jGrid < nxGrids; jGrid++) {
            dof = jGrid + iGrid * nxGrids;
            if (iGrid == 0 || iGrid == nxGrids - 1 || jGrid == 0 || jGrid == nyGrids-1) {
                initX[dof] = 0.0f;
            }
            else {
                initX[dof] = 1.0f; 
            }
            rhs[dof] = 1.0f;
        }
    }

    float * matrixElements = new float[5];
    matrixElements[0] = -1.0f / (dy * dy);
    matrixElements[1] = -1.0f / (dx * dx);
    matrixElements[2] = 2.0f / (dx * dx) + 2.0f / (dy * dy);
    matrixElements[3] = -1.0f / (dx * dx);
    matrixElements[4] = -1.0f / (dy * dy);

    // solutionCPU = iterativeGpu(initX, rhs, matrixElements, nxGrids, nyGrids, nIters);  
    // solutionGPU = iterativeGpu(initX, rhs, matrixElements, nxGrids, nyGrids, nIters);  
    solutionDDGPU = iterativeGpuSwept(initX, rhs, matrixElements, nxGrids, nyGrids, cycles, num_JacobiIters, threadsPerBlock, method, subdomainLength);  
 
    // PRINT RESULTS
/*    for (int iGrid = 0; iGrid < nDofs; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid <<  " Before " << initX[iGrid] << " After " << x0Cpu[iGrid] << std::endl;
    }
*/
    // print2DSolution(solutionDDGPU, nxGrids, nyGrids);    

    // COMPUTE RESIDUAL
    float residual = Residual(solutionDDGPU, rhs, matrixElements, nxGrids, nyGrids);

    // PRINT RESIDUAL
    printf("The residual is %f\n", residual);

    // CLEAN UP
    delete[] initX;
    delete[] rhs;
    delete[] solutionDDGPU;

    return 0;
}



