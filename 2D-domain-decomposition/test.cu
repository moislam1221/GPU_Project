#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

#include "iterative-2D-domain-decomposition-modular.h"

int main(int argc, char *argv[])
{
    // STATUS: 8 BY 8 WITH 4 BY 4 SUBDOMAINS WORKS 
  
    // TRY 12 BY 12 WITH 4 BY 4 SUBDOMAINS 


    // INPUTS
    int nxGrids = 64;
    int nyGrids = 64;
    int subdomainLength = 32; // 4
    int threadsPerBlock = 32; // 4

    int cycles = 100;
    int num_JacobiIters = 100; //atoi(argv[2]);

    printf("Cycles: %d, Jacobi Iterations: %d\n", cycles, num_JacobiIters);

    int method = 0;
   
    float dx = 1.0/ (nxGrids - 1);
    float dy = 1.0/ (nyGrids - 1);
    int nDofs = nxGrids * nyGrids;
    
    // INITIALIZATION
    float * initX = new float[nDofs];
    float * rhs = new float[nDofs];
    float * x0Cpu = new float[nDofs];

    // INITIAL SOLUTION
    int dof;
    for (int iGrid = 0; iGrid < nyGrids; iGrid++) { 
        for (int jGrid = 0; jGrid < nxGrids; jGrid++) {
            dof = jGrid + iGrid * nxGrids;
            if (iGrid == 0 || iGrid == nxGrids - 1 || jGrid == 0 || jGrid == nyGrids-1) {
                initX[dof] = 0.0f;
            }
            else {
                initX[dof] = 1.0f; 
            }
            rhs[dof] = 1.0f;
        }
    }
    // print2DSolution(initX, nxGrids, nyGrids);    

    float * matrixElements = new float[5];
    matrixElements[0] = -1.0f / (dy * dy);
    matrixElements[1] = -1.0f / (dx * dx);
    matrixElements[2] = 2.0f / (dx * dx) + 2.0f / (dy * dy);
    matrixElements[3] = -1.0f / (dx * dx);
    matrixElements[4] = -1.0f / (dy * dy);

    x0Cpu = iterativeGpuSwept(initX, rhs, matrixElements, nxGrids, nyGrids, cycles, num_JacobiIters, threadsPerBlock, method, subdomainLength);  
 
    // PRINT RESULTS
/*    for (int iGrid = 0; iGrid < nDofs; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid <<  " Before " << initX[iGrid] << " After " << x0Cpu[iGrid] << std::endl;
    }
*/
    print2DSolution(x0Cpu, nxGrids, nyGrids);    

    // COMPUTE RESIDUAL
    float residual = Residual(x0Cpu, rhs, matrixElements, nxGrids, nyGrids);

    // PRINT RESIDUAL
    printf("The residual is %f\n", residual);

    // CLEAN UP
    delete[] initX;
    delete[] rhs;
    delete[] x0Cpu;

    return 0;
}



