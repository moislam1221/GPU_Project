#include "hip/hip_runtime.h"
#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

// HEADER FILES
#include "helper.h"
#include "iterative-methods.h"
#include "iterative-1D-cpu.h"
#include "iterative-1D-gpu.h"
#include "iterative-1D-rectangular.h"
#include "iterative-1D-rectangular-multiple.h"

#define PI 3.14159265358979323

int main(int argc, char *argv[])
{
    // INPUTS
    const int nGrids = atoi(argv[1]); 
    const int threadsPerBlock = atoi(argv[2]); 
    const int nInnerUpdates = atoi(argv[3]);
    const int TOL = atoi(argv[4]);
    int nJacobiCpuIterations;
    int nJacobiGpuIterations;
    int nJacobiCycles;

    // INITIALIZE ARRAYS
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    float * leftMatrix = new float[nGrids];
    float * centerMatrix = new float[nGrids];
    float * rightMatrix = new float[nGrids];
    float dx = 1.0f / (nGrids + 1);
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
        leftMatrix[iGrid] = -1.0f / (dx * dx);
        centerMatrix[iGrid] = 2.0f / (dx * dx);
        rightMatrix[iGrid] = -1.0f / (dx * dx);
    }

    // OBTAIN NUMBER OF ITERATIONS NECESSARY TO ACHIEVE TOLERANCE FOR EACH METHOD
    nJacobiCpuIterations = jacobiCpuIterationCount(initX, rhs, leftMatrix, centerMatrix,
                                    rightMatrix, nGrids, TOL);
    nJacobiGpuIterations = jacobiGpuClassicIterationCount(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids, TOL, threadsPerBlock);
    nJacobiCycles = jacobiGpuRectangularIterationCount(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, TOL, nInnerUpdates);
    
    // CPU - JACOBI
    clock_t cpuJacobiStartTime = clock();
    float * solutionJacobiCpu = jacobiCpu(initX, rhs, leftMatrix, centerMatrix,
                                    rightMatrix, nGrids, nJacobiCpuIterations);
    clock_t cpuJacobiEndTime = clock();
    float cpuJacobiTime = (cpuJacobiEndTime - cpuJacobiStartTime) / (float) CLOCKS_PER_SEC;
    cpuJacobiTime = cpuJacobiTime * (1e3); // Convert to ms

    // GPU - JACOBI
    hipEvent_t startJacobiGpu, stopJacobiGpu;
    float gpuJacobiTime;
    hipEventCreate( &startJacobiGpu );
    hipEventCreate( &stopJacobiGpu );
    hipEventRecord(startJacobiGpu, 0);
    float * solutionJacobiGpu = jacobiGpuClassic(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids, nJacobiGpuIterations, threadsPerBlock);
    hipEventRecord(stopJacobiGpu, 0);
    hipEventSynchronize(stopJacobiGpu);
    hipEventElapsedTime(&gpuJacobiTime, startJacobiGpu, stopJacobiGpu);

    // RECTANGULAR METHOD - JACOBI
    hipEvent_t startJacobiGpuRectangular, stopJacobiGpuRectangular;
    float gpuJacobiRectangularTime;
    hipEventCreate( &startJacobiGpuRectangular );
    hipEventCreate( &stopJacobiGpuRectangular );
    hipEventRecord( startJacobiGpuRectangular, 0);
    float * solutionJacobiGpuRectangular = jacobiGpuRectangular(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, nJacobiCycles, nInnerUpdates);
    hipEventRecord(stopJacobiGpuRectangular, 0);
    hipEventSynchronize(stopJacobiGpuRectangular);
    hipEventElapsedTime(&gpuJacobiRectangularTime, startJacobiGpuRectangular, stopJacobiGpuRectangular);

    // COMPUTE TIME FACTORS   
    float cpuToGpu = cpuJacobiTime / gpuJacobiTime;
    float gpuToRectangular = gpuJacobiTime / gpuJacobiRectangularTime;
    float cpuToRectangular = cpuJacobiTime / gpuJacobiRectangularTime;
/*
    // PRINT SOLUTION
    for (int i = 0; i < nGrids; i++) {
        printf("Grid %d = %f %f %f %f %f %f\n", i, solutionJacobiCpu[i], solutionJacobiGpu[i], solutionJacobiGpuRectangular[i]);
    }
*/
    
    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of grid points: %d\n", nGrids);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("Number of Cycles of Jacobi Rectangular performed: %d\n", nJacobiCycles);
    printf("CPU -> GPU Speedup Factor is %f\n", cpuToGpu);
    printf("GPU -> GPU Rectangular Speedup Factor is %f\n", gpuToRectangular);
    printf("CPU -> GPU Rectangular Speedup Factor is %f\n", cpuToRectangular);
    printf("======================================================\n");
    
    // Print out number of iterations needed for each method
    printf("Number of Iterations needed for Jacobi CPU: %d \n", nJacobiCpuIterations);
    printf("Number of Iterations needed for Jacobi GPU: %d \n", nJacobiGpuIterations);
    printf("Number of Cycles needed for Jacobi GPU Rectangular: %d (with %d inner updates) \n", nJacobiCycles, nInnerUpdates);
    printf("======================================================\n");
    
    // Print out time for cpu, classic gpu, and swept gpu approaches
    printf("Time needed for the Jacobi CPU: %f ms\n", cpuJacobiTime);
    printf("Time needed for the Jacobi GPU: %f ms\n", gpuJacobiTime);
    printf("Time needed for the Jacobi GPU Rectangular method: %f ms\n", gpuJacobiRectangularTime);
    printf("======================================================\n");

    // Compute the residual of the resulting solution (|b-Ax|)
    float residualJacobiCpu = Residual(solutionJacobiCpu, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    float residualJacobiGpu = Residual(solutionJacobiGpu, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    float residualJacobiGpuRectangular = Residual(solutionJacobiGpuRectangular, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    printf("Residual of the Jacobi CPU solution is %f\n", residualJacobiCpu);
    printf("Residual of the Jacobi GPU solution is %f\n", residualJacobiGpu);
    printf("Residual of the Jacobi Rectangular solution is %f\n", residualJacobiGpuRectangular);

/*    for (int i = 0; i < nGrids; i++) {
        if (i == 0 || i == nGrids-1) {
            assert(solutionGpuRectangular[i] == 0.0);
        }
        else {
            assert(solutionGpuRectangular[i] == (float)(cycles * nIterations + 1.0));
        }
    }   
*/

/*    // Print out time for cpu, classic gpu, and swept gpu approaches
    float cpuTimePerIteration = (cpuTime / nIters) * 1e3;
    float classicTimePerIteration = gpuTime / nIters;
    float sweptTimePerIteration = timeSwept / nIters;
    float timeMultiplier = classicTimePerIteration / sweptTimePerIteration;
    printf("Time needed for the CPU (per iteration): %f ms\n", cpuTimePerIteration);
    printf("Time needed for the Classic GPU (per iteration) is %f ms\n", classicTimePerIteration);
    printf("Time needed for the Swept GPU (per iteration): %f ms\n", sweptTimePerIteration);
*/

    // Write results to file
    std::ofstream results;
    results.open("results-jac.txt", std::ios_base::app);
    results << nGrids << " " << threadsPerBlock << " " << TOL << " " << cpuJacobiTime << " " << gpuJacobiTime << " " << gpuJacobiRectangularTime << " " << "\n";
    results.close();

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] leftMatrix;
    delete[] centerMatrix;
    delete[] rightMatrix;
    delete[] solutionJacobiCpu;
    delete[] solutionJacobiGpu;
    delete[] solutionJacobiGpuRectangular;
    
    return 0;
}
