#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

// HEADER FILES
#include "helper.h"
#include "iterative-methods.h"
#include "iterative-1D-cpu.h"
#include "iterative-1D-gpu.h"
#include "iterative-1D-rectangular.h"
#include "iterative-1D-rectangular-multiple.h"

int main(int argc, char *argv[])
{
    // INPUTS
    const int nGrids = atoi(argv[1]); 
    const int threadsPerBlock = atoi(argv[2]); 
    const int nInnerUpdates = atoi(argv[3]);
    const int pMultiple = atoi(argv[4]);
    const int TOL = atoi(argv[5]);
    method_type method = JACOBI;
    int nCycles;
    int nCyclesMultiple;

    // INITIALIZE ARRAYS
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    float * leftMatrix = new float[nGrids];
    float * centerMatrix = new float[nGrids];
    float * rightMatrix = new float[nGrids];
    float dx = 1.0f / (nGrids + 1);
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
        leftMatrix[iGrid] = -1.0f / (dx * dx);
        centerMatrix[iGrid] = 2.0f / (dx * dx);
        rightMatrix[iGrid] = -1.0f / (dx * dx);
    }

    // OBTAIN NUMBER OF ITERATIONS NECESSARY TO ACHIEVE TOLERANCE
    printf("Hello\n");
    nCycles = iterativeGpuRectangularIterationCount(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, TOL, nInnerUpdates, method);
    printf("Hello\n");
    nCyclesMultiple = iterativeGpuRectangularMultipleIterationCount(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, TOL, nInnerUpdates, method, pMultiple);
    printf("Hello\n");

    // RECTANGULAR METHOD
    hipEvent_t startGpuRectangular, stopGpuRectangular;
    float gpuRectangularTime;
    hipEventCreate( &startGpuRectangular );
    hipEventCreate( &stopGpuRectangular );
    hipEventRecord( startGpuRectangular, 0);
    float * solutionGpuRectangular = iterativeGpuRectangular(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, nCycles, nInnerUpdates, method);
    hipEventRecord(stopGpuRectangular, 0);
    hipEventSynchronize(stopGpuRectangular);
    hipEventElapsedTime(&gpuRectangularTime, startGpuRectangular, stopGpuRectangular);
    
    // RECTANGULAR MULTIPLE METHOD
    hipEvent_t startGpuRectangularMultiple, stopGpuRectangularMultiple;
    float gpuRectangularMultipleTime;
    hipEventCreate( &startGpuRectangularMultiple );
    hipEventCreate( &stopGpuRectangularMultiple );
    hipEventRecord( startGpuRectangularMultiple, 0);
    float * solutionGpuRectangularMultiple = iterativeGpuRectangularMultiple(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, nCyclesMultiple, nInnerUpdates, method, pMultiple);
    hipEventRecord(stopGpuRectangularMultiple, 0);
    hipEventSynchronize(stopGpuRectangularMultiple);
    hipEventElapsedTime(&gpuRectangularMultipleTime, startGpuRectangularMultiple, stopGpuRectangularMultiple);

    // PRINT SOLUTION
    for (int i = 0; i < nGrids; i++) {
        printf("Grid %d = %f %f\n", i, solutionGpuRectangular[i], solutionGpuRectangularMultiple[i]);
    }
    
    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of grid points: %d\n", nGrids);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("Method used: %d\n", method);
    printf("Number of Cycles of Rectangular performed: %d\n", nCycles);
    printf("======================================================\n");
    printf("\n");
    
    // Print out number of iterations needed for each method
    printf("Number of Cycles needed for GPU Rectangular: %d (with %d inner updates) \n", nCycles, nInnerUpdates);
    printf("Number of Cycles needed for GPU Rectangular Multiple: %d (with %d inner updates and %d points per thread) \n", nCycles, nInnerUpdates, pMultiple);
    printf("======================================================\n");
    printf("\n");
    
    // Print out time for cpu, classic gpu, and swept gpu approaches
    printf("Time needed for the GPU Rectangular method: %f ms\n", gpuRectangularTime);
    printf("Time needed for the GPU Rectangular Multiple method: %f ms\n", gpuRectangularMultipleTime);
    printf("======================================================\n");
    printf("\n");


    // Compute the residual of the resulting solution (|b-Ax|)
    float residualGpuRectangular = Residual(solutionGpuRectangular, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    float residualGpuRectangularMultiple = Residual(solutionGpuRectangularMultiple, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    printf("Residual of the Rectangular solution is %f\n", residualGpuRectangular);
    printf("Residual of the Rectangular Multiple solution is %f\n", residualGpuRectangularMultiple);

/*    for (int i = 0; i < nGrids; i++) {
        if (i == 0 || i == nGrids-1) {
            assert(solutionGpuRectangular[i] == 0.0);
        }
        else {
            assert(solutionGpuRectangular[i] == (float)(cycles * nIterations + 1.0));
        }
    }   
*/

/*    // Print out time for cpu, classic gpu, and swept gpu approaches
    float cpuTimePerIteration = (cpuTime / nIters) * 1e3;
    float classicTimePerIteration = gpuTime / nIters;
    float sweptTimePerIteration = timeSwept / nIters;
    float timeMultiplier = classicTimePerIteration / sweptTimePerIteration;
    printf("Time needed for the CPU (per iteration): %f ms\n", cpuTimePerIteration);
    printf("Time needed for the Classic GPU (per iteration) is %f ms\n", classicTimePerIteration);
    printf("Time needed for the Swept GPU (per iteration): %f ms\n", sweptTimePerIteration);
*/

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] leftMatrix;
    delete[] centerMatrix;
    delete[] rightMatrix;
    delete[] solutionGpuRectangular;
    delete[] solutionGpuRectangularMultiple;

    return 0;
}
