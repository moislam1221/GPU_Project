#include "hip/hip_runtime.h"
#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

// HEADER FILES
#include "helper.h"
#include "iterative-methods.h"
#include "iterative-1D-cpu.h"
#include "iterative-1D-gpu.h"
#include "iterative-1D-rectangular.h"
#include "iterative-1D-rectangular-multiple.h"

int main(int argc, char *argv[])
{
    // INPUTS
/*    const int nGrids = atoi(argv[1]); 
    const int threadsPerBlock = atoi(argv[2]); 
    const int cycles = atoi(argv[3]);
    const int nIterations = atoi(argv[4]);
    const int nCpuIterations = 2; //atoi(argv[3]);
    const int nGpuIterations = atoi(argv[5]);
    method_type method = JACOBI;
*/
    // INPUTS
    const int nGrids = atoi(argv[1]); 
    const int threadsPerBlock = atoi(argv[2]); 
    const int nInnerUpdates = atoi(argv[3]);
    const int TOL = atoi(argv[4]);
    method_type method = JACOBI;
    int nCpuIterations;
    int nGpuIterations;
    int nCycles;

    // INITIALIZE ARRAYS
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    float * leftMatrix = new float[nGrids];
    float * centerMatrix = new float[nGrids];
    float * rightMatrix = new float[nGrids];
    float dx = 1.0f / (nGrids + 1);
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
        leftMatrix[iGrid] = -1.0f / (dx * dx);
        centerMatrix[iGrid] = 2.0f / (dx * dx);
        rightMatrix[iGrid] = -1.0f / (dx * dx);
    }

    // OBTAIN NUMBER OF ITERATIONS NECESSARY TO ACHIEVE TOLERANCE
    nCpuIterations = iterativeCpuIterationCount(initX, rhs, leftMatrix, centerMatrix,
                                    rightMatrix, nGrids, TOL, method);
    nGpuIterations = iterativeGpuClassicIterationCount(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids, TOL, threadsPerBlock, method);
    nCycles = iterativeGpuRectangularIterationCount(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, TOL, nInnerUpdates, method);

    // CPU
    clock_t cpuStartTime = clock();
    float * solutionCpu = iterativeCpu(initX, rhs, leftMatrix, centerMatrix,
                                    rightMatrix, nGrids, nCpuIterations, method);
    clock_t cpuEndTime = clock();
    double cpuTime = (cpuEndTime - cpuStartTime) / (float) CLOCKS_PER_SEC;
    cpuTime = cpuTime * (1e3); // Convert to ms

    // GPU
    hipEvent_t startGpu, stopGpu;
    float gpuTime;
    hipEventCreate( &startGpu );
    hipEventCreate( &stopGpu );
    hipEventRecord(startGpu, 0);
    float * solutionGpu = iterativeGpuClassic(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids, nGpuIterations, threadsPerBlock, method);
    hipEventRecord(stopGpu, 0);
    hipEventSynchronize(stopGpu);
    hipEventElapsedTime(&gpuTime, startGpu, stopGpu);

    // RECTANGULAR METHOD
    hipEvent_t startGpuRectangular, stopGpuRectangular;
    float gpuRectangularTime;
    hipEventCreate( &startGpuRectangular );
    hipEventCreate( &stopGpuRectangular );
    hipEventRecord( startGpuRectangular, 0);
    float * solutionGpuRectangular = iterativeGpuRectangular(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, nCycles, nInnerUpdates, method);
    hipEventRecord(stopGpuRectangular, 0);
    hipEventSynchronize(stopGpuRectangular);
    hipEventElapsedTime(&gpuRectangularTime, startGpuRectangular, stopGpuRectangular);
/*    
    // RECTANGULAR MULTIPLE METHOD
    hipEvent_t startGpuRectangularMultiple, stopGpuRectangularMultiple;
    float gpuRectangularMultipleTime;
    hipEventCreate( &startGpuRectangularMultiple );
    hipEventCreate( &stopGpuRectangularMultiple );
    hipEventRecord( startGpuRectangularMultiple, 0);
    float * solutionGpuRectangularMultiple = iterativeGpuRectangularMultiple(initX, rhs, leftMatrix,
            centerMatrix, rightMatrix, nGrids,  threadsPerBlock, cycles, nIterations, method, 10);
    hipEventRecord(stopGpuRectangularMultiple, 0);
    hipEventSynchronize(stopGpuRectangularMultiple);
    hipEventElapsedTime(&gpuRectangularMultipleTime, startGpuRectangularMultiple, stopGpuRectangularMultiple);
*/
    // PRINT SOLUTION
    for (int i = 0; i < nGrids; i++) {
        printf("Grid %d = %f %f %f\n", i, solutionCpu[i], solutionGpu[i], solutionGpuRectangular[i]);
    }
    
    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of grid points: %d\n", nGrids);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("Method used: %d\n", method);
    printf("Number of Cycles of Rectangular performed: %d\n", nCycles);
    printf("======================================================\n");
    printf("\n");
    
    // Print out number of iterations needed for each method
    printf("Number of Iterations needed for CPU: %d \n", nCpuIterations);
    printf("Number of Iterations needed for GPU: %d \n", nGpuIterations);
    printf("Number of Cycles needed for GPU Rectangular: %d (with %d inner updates) \n", nCycles, nInnerUpdates);
    //printf("Time needed for the GPU Rectangular Multiple method: %f ms\n", gpuRectangularMultipleTime);
    printf("======================================================\n");
    printf("\n");
    
    // Print out time for cpu, classic gpu, and swept gpu approaches
    printf("Time needed for the CPU: %f ms\n", cpuTime);
    printf("Time needed for the GPU: %f ms\n", gpuTime);
    printf("Time needed for the GPU Rectangular method: %f ms\n", gpuRectangularTime);
    //printf("Time needed for the GPU Rectangular Multiple method: %f ms\n", gpuRectangularMultipleTime);
    printf("======================================================\n");
    printf("\n");


    // Compute the residual of the resulting solution (|b-Ax|)
    float residualCpu = Residual(solutionCpu, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    float residualGpu = Residual(solutionGpu, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    float residualGpuRectangular = Residual(solutionGpuRectangular, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    // float residualGpuRectangularMultiple = Residual(solutionGpuRectangularMultiple, rhs, leftMatrix, centerMatrix, rightMatrix, nGrids);
    printf("Residual of the CPU solution is %f\n", residualCpu);
    printf("Residual of the GPU solution is %f\n", residualGpu);
    printf("Residual of the Rectangular solution is %f\n", residualGpuRectangular);
    // printf("Residual of the Rectangular Multiple solution is %f\n", residualGpuRectangularMultiple);

/*    for (int i = 0; i < nGrids; i++) {
        if (i == 0 || i == nGrids-1) {
            assert(solutionGpuRectangular[i] == 0.0);
        }
        else {
            assert(solutionGpuRectangular[i] == (float)(cycles * nIterations + 1.0));
        }
    }   
*/

/*    // Print out time for cpu, classic gpu, and swept gpu approaches
    float cpuTimePerIteration = (cpuTime / nIters) * 1e3;
    float classicTimePerIteration = gpuTime / nIters;
    float sweptTimePerIteration = timeSwept / nIters;
    float timeMultiplier = classicTimePerIteration / sweptTimePerIteration;
    printf("Time needed for the CPU (per iteration): %f ms\n", cpuTimePerIteration);
    printf("Time needed for the Classic GPU (per iteration) is %f ms\n", classicTimePerIteration);
    printf("Time needed for the Swept GPU (per iteration): %f ms\n", sweptTimePerIteration);
*/

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] leftMatrix;
    delete[] centerMatrix;
    delete[] rightMatrix;
    delete[] solutionGpu;
    delete[] solutionGpuRectangular;
    //delete[] solutionGpuRectangularMultiple;

    return 0;
}
