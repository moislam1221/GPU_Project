#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

#include "original-strided.h"

int main()
{
    // INPUTS
    int nxGrids = 8;
    int nyGrids = 8;
    int subdomainLength = 4;
    int threadsPerBlock = 2;
   
    // SETTING GRID, BLOCK, THREAD INFORMATION 
    int nxBlocks = nxGrids / subdomainLength;
    int nyBlocks = nyGrids / subdomainLength;
    dim3 grid(nxBlocks, nyBlocks);
    dim3 block(threadsPerBlock, threadsPerBlock);
    int sharedMemory = subdomainLength * subdomainLength * 2 * sizeof(double);
    
    // PANEL ARRAY SIZES
    int elemPerBlock = subdomainLength * subdomainLength;
    int numBridgeElemPerBlock = elemPerBlock / 2;
    int numBridgeElemTotal = nxBlocks * nyBlocks * numBridgeElemPerBlock;

    // OTHER PARAMETERS
    int dx = 1.0/ (nxGrids + 1);
    int dy = 1.0/ (nyGrids + 1);
    int nDofs = nxGrids * nyGrids;
    
    // INT AND POINTER FOR UNUSED PARAMETERS    
    int notUsedInt;
    double * notUsed;

    // INITIALIZATION
    double * initX = new double[nDofs];
    double * x0Cpu = new double[nDofs];
    double * xLeftCpu = new double[numBridgeElemTotal];
    double * xRightCpu = new double[numBridgeElemTotal];
    double * xTopCpu = new double[numBridgeElemTotal];
    double * xBottomCpu = new double[numBridgeElemTotal];

    // INITIAL SOLUTION
    for (int iGrid = 0; iGrid < nDofs; iGrid++) 
    {
        initX[iGrid] = iGrid;
//          initX[iGrid] = 1.0;
    }

    // ALLOCATE SOLUTION MEMORY - CPU AND GPU
    double * x0Gpu;
    hipMalloc(&x0Gpu, sizeof(double) * (nDofs));
    hipMemcpy(x0Gpu, initX, sizeof(double) * nDofs, hipMemcpyHostToDevice);
    
    // ALLOCATE PANEL ARRAY MEMORY
    double *xLeftGpu, *xRightGpu, *xTopGpu, *xBottomGpu;
    hipMalloc(&xLeftGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xRightGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xTopGpu, sizeof(double) * numBridgeElemTotal);
    hipMalloc(&xBottomGpu, sizeof(double) * numBridgeElemTotal);
    
    // PRINT TO SCREEN
    std::cout << "\n" << "NOW APPLYING FIRST UPDATE" << "\n" << std::endl;
    
    // APPLY METHOD TO ADVANCE POINTS (NO SHIFT)
    _iterativeGpuOriginal <<<grid, block, sharedMemory>>> (xLeftGpu, xRightGpu, x0Gpu, notUsed,
		   						 notUsed, notUsed, notUsed, notUsed, notUsed, nxGrids, nyGrids, notUsedInt, subdomainLength);

    // COPY TO CPU 
    hipMemcpy(xLeftCpu, xLeftGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xRightCpu, xRightGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    
    // PRINT RESULTS
    for (int iGrid = 0; iGrid < numBridgeElemTotal; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid << " xLeft " << xLeftCpu[iGrid] << " xRight " << xRightCpu[iGrid] << std::endl;
    }

    // PRINT TO SCREEN
    std::cout << "\n" << "NOW APPLYING HORIZONTAL UPDATE" << "\n" << std::endl;

    // APPLY HORIZONTAL SHIFT
    _iterativeGpuHorizontalShift <<<grid, block, sharedMemory>>> (xLeftGpu, xRightGpu, xTopGpu, xBottomGpu, x0Gpu, notUsed,
		   						 notUsed, notUsed, notUsed, notUsed, notUsed, nxGrids, nyGrids, notUsedInt, subdomainLength);
   
    // COPY TO CPU 
    hipMemcpy(xLeftCpu, xLeftGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xRightCpu, xRightGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xTopCpu, xTopGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xBottomCpu, xBottomGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    
    // PRINT RESULTS
    for (int iGrid = 0; iGrid < numBridgeElemTotal; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid << " xTop " << xTopCpu[iGrid] << " xBottom " << xBottomCpu[iGrid] << std::endl;
    }


    // PRINT TO SCREEN
//    std::cout << "\n" << "NOW APPLYING VERTICAL AND HORIZONTAL SHIFT METHOD" << "\n" << std::endl;

    // APPLY VERTICAL SHIFT
//    _iterativeGpuVerticalandHorizontalShift <<<grid, block, sharedMemory>>> (xLeftGpu, xRightGpu, xTopGpu, xBottomGpu, x0Gpu, notUsed,
//		   						 notUsed, notUsed, notUsed, notUsed, notUsed, nxGrids, nyGrids, notUsedInt);
   
/*    // COPY TO CPU 
    hipMemcpy(xLeftCpu, xLeftGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xRightCpu, xRightGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xTopCpu, xTopGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xBottomCpu, xBottomGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    
    // PRINT RESULTS
    for (int iGrid = 0; iGrid < numBridgeElemTotal; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid << " xLeft " << xLeftCpu[iGrid] << " xRight " << xRightCpu[iGrid] << " xTop " << xTopCpu[iGrid] << " xBottom " << xBottomCpu[iGrid] << std::endl;
    }
*/
    // PRINT TO SCREEN
//    std::cout << "\n" << "NOW APPLYING PURELY VERTICAL SHIFT METHOD" << "\n" << std::endl;

    // APPLY VERTICAL SHIFT
//    _iterativeGpuVerticalShift <<<grid, block, sharedMemory>>> (xLeftGpu, xRightGpu, xTopGpu, xBottomGpu, x0Gpu, notUsed,
//		   						 notUsed, notUsed, notUsed, notUsed, notUsed, nxGrids, nyGrids, notUsedInt);
/*   
    // COPY TO CPU 
    hipMemcpy(xLeftCpu, xLeftGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xRightCpu, xRightGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xTopCpu, xTopGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    hipMemcpy(xBottomCpu, xBottomGpu, sizeof(double) * numBridgeElemTotal, hipMemcpyDeviceToHost);
    
    // PRINT RESULTS
    for (int iGrid = 0; iGrid < numBridgeElemTotal; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid <<  " xTop " << xTopCpu[iGrid] << " xBottom " << xBottomCpu[iGrid] << std::endl;
    }
*/    
    // PRINT TO SCREEN
//    std::cout << "\n" << "NOW APPLYING FINAL METHOD" << "\n" << std::endl;

    // APPLY FINAL STEP
//    _finalSolution <<<grid, block, sharedMemory>>>(xTopGpu, xBottomGpu, x0Gpu, nxGrids);
    
    // COPY TO CPU 
//    hipMemcpy(x0Cpu, x0Gpu, sizeof(double) * nDofs, hipMemcpyDeviceToHost);
    
    // PRINT RESULTS
/*    for (int iGrid = 0; iGrid < nDofs; iGrid++) 
    {
        std::cout << "Grid Point " << iGrid <<  " x0Cpu " << x0Cpu[iGrid]  << std::endl;
    }
*/

    // CLEAN UP
    hipFree(xLeftGpu);
    hipFree(xRightGpu);
    hipFree(xTopGpu);
    hipFree(xBottomGpu);
    delete[] initX;
    delete[] xLeftCpu;
    delete[] xRightCpu;
    delete[] xTopCpu;
    delete[] xBottomCpu;    
}



